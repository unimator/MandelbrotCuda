﻿#include <windows.h>
#include <cstdint>
#include <cstdio>
#include "ComplexNumber.h"
#include "Mandelbrot.h"

template <typename TNumber>
uint32_t* CalculateMandelbrot(TNumber re, TNumber im, TNumber scale_x, TNumber scale_y, size_t width, size_t height, uint32_t max_iterations);

BOOL WINAPI DllMain(
	_In_ HINSTANCE hinstDLL,
	_In_ DWORD fdwReason,
	_In_ LPVOID lpvReserved
)
{
	return true;
}

extern "C" {

	__declspec(dllexport)
	void FreeMemory(void *ptr)
	{
		if(ptr)
		{
			free(ptr);
		}
	}

	__declspec(dllexport)
	uint32_t* CalculateMandelbrotDouble(const double re, const double im, const double scale_x, const double scale_y, const size_t width,
	                                           const size_t height, const uint32_t max_iterations)
	{
		return CalculateMandelbrot<double>(re, im, scale_x, scale_y, width, height, max_iterations);
	}

	__declspec(dllexport)
	uint32_t* CalculateMandelbrotFloat(const float re, const float im, const float scale_x, const double scale_y, const size_t width,
	                                          const size_t height, const uint32_t max_iterations)
	{
		return CalculateMandelbrot<float>(re, im, scale_x, scale_y, width, height, max_iterations);
	}
}

template <typename TNumber>
uint32_t* CalculateMandelbrot(TNumber re, TNumber im, TNumber scale_x, TNumber scale_y, size_t width, size_t height,
                                     uint32_t max_iterations)
{
#if _DEBUG
	printf("CalculateMandelbrot(%llf, %llf, %llf, %llf, %u, %u, %u)\n", re, im, scale_x, scale_y, width, height, max_iterations);
#endif
	auto origin = ComplexNumber<TNumber>(re, im);
	auto mandelbrot = Mandelbrot<TNumber>(origin, scale_x, scale_y, width, height, max_iterations);
	return mandelbrot.GetData();
}